#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "relax.cu"
#include <iostream>
#include <time.h>

using namespace std;

#define Delta 10

char* dir="graph.txt";

int cal_shortest_path();
void bellman_ford(graph &Graph, int &total_Teps);
int deltaHybrid();

int main(void){
    clock_t start = clock();
    int TEPS = deltaHybrid();
    cout<<"TEPS: "<<TEPS<<endl;
    
    /*graph Graph(dir);
    bellman_ford(Graph);*/
    int time = ((double)clock()-start/CLOCKS_PER_SEC);
    cout<<"Tiempo transcurrido: "<<time<<endl;
    cout<<"GTEPS: "<<(double)TEPS/(double)time<<endl;
}

void printResult(graph &graph_instance){
    for(int i=0; i<graph_instance.vertex_size; ++i){
        cout<<i<<" "<<graph_instance.global_vertex[i].pre_vertex<<" "<<graph_instance.global_vertex[i].dist<<endl;
    }
}

int cal_shortest_path(){
    int num_block = 8;
    int num_threads = 32;
    dim3 dg(num_block, 1, 1);
    dim3 db(num_threads, 1, 1);
    graph graph_instance(dir,Delta);
    int* TEPS_GPU,*TEPS=new int[num_block*num_threads];
    for(int i=0; i<num_threads*num_block; ++i)
        TEPS[i]=0;

    hipSetDevice(0);

    hipMalloc((void **)&TEPS_GPU,num_block*num_threads*sizeof(int));
    hipMemcpy(TEPS_GPU,TEPS,num_block*num_threads*sizeof(int),hipMemcpyHostToDevice);


    hipMalloc((void **)&graph_instance.gpu_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex));
    hipMalloc((void**)&graph_instance.gpu_edge,graph_instance.edges_size*sizeof(graph::edge));

    hipMalloc((void**)&graph_instance.gpu_used_result_buf,MAX_RESULT_SIZE * sizeof(graph::gpuResult));  

    hipMemcpy(graph_instance.gpu_vertex,graph_instance.global_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex),hipMemcpyHostToDevice);
    hipMemcpy(graph_instance.gpu_edge,graph_instance.global_edge,graph_instance.edges_size*sizeof(graph::edge), hipMemcpyHostToDevice);

    hipMemcpy(graph_instance.gpu_used_result_buf,graph_instance.gpu_result_buf,sizeof(graph::gpuResult) * MAX_RESULT_SIZE, hipMemcpyHostToDevice);

    hipMalloc((void**)&graph_instance.vertex_buf_ptr, MAX_BUCKET_SIZE);


    int min,result_count=0;
    int* temp_vertex_array;
    temp_vertex_array = (int*)malloc(sizeof(int) * MAX_BUCKET_SIZE);
    int timeInBucket=0;
    int cantidad=0;
    while(!graph_instance.is_all_bucket_empty()){
        cantidad++;
        min = graph_instance.min_no_empty_bucket(); 
        int count = graph_instance.bucket_set_to_array(min, temp_vertex_array);

        for(int i=count; i< MAX_BUCKET_SIZE; i++)
                temp_vertex_array[i] = 0;
	
        for(int i = 0; i < MAX_BUCKET_SIZE; i++){
            if(temp_vertex_array[i] != 0){
                graph_instance.bucket_array[min].erase(temp_vertex_array[i]);
            }
        }
		
        hipMemcpy(graph_instance.vertex_buf_ptr,temp_vertex_array, MAX_BUCKET_SIZE,hipMemcpyHostToDevice);
       relax_all<<<num_block,num_threads>>>(graph_instance.vertex_buf_ptr,
               graph_instance.gpu_vertex,graph_instance.gpu_edge,graph_instance.gpu_result_buf,
               graph_instance.gpu_used_result_buf,TEPS_GPU);
       hipMemcpy(graph_instance.gpu_result_buf,graph_instance.gpu_used_result_buf,
                   sizeof(graph::gpuResult)*num_threads*num_block, hipMemcpyDeviceToHost); 

       result_count = 0;
       clock_t start = clock();
       while(1){
            if(result_count >= MAX_BUCKET_SIZE){
                break;
            }
            if(graph_instance.gpu_result_buf[result_count].index == 0){
                result_count++;
                continue;
             }

             
            int old_index = graph_instance.gpu_result_buf[result_count].old_distance / graph_instance.delta;
            int new_index = graph_instance.gpu_result_buf[result_count].new_distance / graph_instance.delta;
            if(graph_instance.gpu_result_buf[result_count].old_distance != MAX_DISTANCE){
                graph_instance.bucket_array[old_index].erase(graph_instance.gpu_result_buf[result_count].index);
            }
                   
            graph_instance.bucket_array[new_index].insert(graph_instance.gpu_result_buf[result_count].index);
            result_count++;
        }
        timeInBucket += ((double)clock()-start/CLOCKS_PER_SEC);
    }
    
    cout<<"Tiempo en bucket: "<<timeInBucket<<endl;
    cout<<"Numero de buckets: "<<cantidad<<endl;
    hipMemcpy(graph_instance.global_vertex,graph_instance.gpu_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex),hipMemcpyDeviceToHost);
    
    printf("over\n");
    
    hipMemcpy(TEPS,TEPS_GPU,num_block*num_threads*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(graph_instance.gpu_vertex);
    hipFree(graph_instance.gpu_edge);
    hipFree(graph_instance.gpu_used_result_buf);  
    hipFree(TEPS_GPU);
    free(temp_vertex_array);
    int total_Teps=0;
    for(int i=0; i<num_block*num_threads; ++i){
        total_Teps+=TEPS[i];
    }
    return total_Teps;
}

int deltaHybrid(){
    int num_block = 8;
    int num_threads = 32;
    dim3 dg(num_block, 1, 1);
    dim3 db(num_threads, 1, 1);
    graph graph_instance(dir,Delta);
    int* TEPS_GPU,*TEPS=new int[num_block*num_threads];
    for(int i=0; i<num_threads*num_block; ++i)
        TEPS[i]=0;

    hipSetDevice(0);

    hipMalloc((void **)&TEPS_GPU,num_block*num_threads*sizeof(int));
    hipMemcpy(TEPS_GPU,TEPS,num_block*num_threads*sizeof(int),hipMemcpyHostToDevice);

     //copy to GPU
    hipMalloc((void **)&graph_instance.gpu_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex));
    hipMalloc((void**)&graph_instance.gpu_edge,graph_instance.edges_size*sizeof(graph::edge));

    //malloc danteng!!
    hipMalloc((void**)&graph_instance.gpu_used_result_buf,MAX_RESULT_SIZE * sizeof(graph::gpuResult));  
    //copy  
    hipMemcpy(graph_instance.gpu_vertex,graph_instance.global_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex),hipMemcpyHostToDevice);
    hipMemcpy(graph_instance.gpu_edge,graph_instance.global_edge,graph_instance.edges_size*sizeof(graph::edge), hipMemcpyHostToDevice);

    hipMemcpy(graph_instance.gpu_used_result_buf,graph_instance.gpu_result_buf,sizeof(graph::gpuResult) * MAX_RESULT_SIZE, hipMemcpyHostToDevice);
    //malloc vertex buffer
    hipMalloc((void**)&graph_instance.vertex_buf_ptr, MAX_BUCKET_SIZE);
    //malloc result buffer

    int min=0,result_count=0;
    int* temp_vertex_array;
    temp_vertex_array = (int*)malloc(sizeof(int) * MAX_BUCKET_SIZE);
    int cantidad=0;
    while(!graph_instance.is_all_bucket_empty() && (float) graph_instance.getFinishedNodes(min)/(float) graph_instance.vertex_size <0.4){
        cantidad++;
        min = graph_instance.min_no_empty_bucket(); 
        int count = graph_instance.bucket_set_to_array(min, temp_vertex_array);

        for(int i=count; i< MAX_BUCKET_SIZE; i++)
                temp_vertex_array[i] = 0;
	
        for(int i = 0; i < MAX_BUCKET_SIZE; i++){
            if(temp_vertex_array[i] != 0){
                graph_instance.bucket_array[min].erase(temp_vertex_array[i]);
            }
        }
		
        hipMemcpy(graph_instance.vertex_buf_ptr,temp_vertex_array, MAX_BUCKET_SIZE,hipMemcpyHostToDevice);

       relax_all<<<num_block,num_threads>>>(graph_instance.vertex_buf_ptr,
               graph_instance.gpu_vertex,graph_instance.gpu_edge,graph_instance.gpu_result_buf,
               graph_instance.gpu_used_result_buf,TEPS_GPU);
       hipMemcpy(graph_instance.gpu_result_buf,graph_instance.gpu_used_result_buf,
                   sizeof(graph::gpuResult)*num_threads*num_block, hipMemcpyDeviceToHost); 

       result_count = 0;
       while(1){
            if(result_count >= MAX_BUCKET_SIZE){
                break;
            }
            if(graph_instance.gpu_result_buf[result_count].index == 0){
                result_count++;
                continue;
             }

             
            int old_index = graph_instance.gpu_result_buf[result_count].old_distance / graph_instance.delta;
            int new_index = graph_instance.gpu_result_buf[result_count].new_distance / graph_instance.delta;
            if(graph_instance.gpu_result_buf[result_count].old_distance != MAX_DISTANCE){
                graph_instance.bucket_array[old_index].erase(graph_instance.gpu_result_buf[result_count].index);
            }
                   
            graph_instance.bucket_array[new_index].insert(graph_instance.gpu_result_buf[result_count].index);
            result_count++;
        }
       
    }
    cout<<"Cantidad de buckets: "<<cantidad<<endl;
    get_result<<<1,1>>>(graph_instance.gpu_vertex,2);
    hipMemcpy(graph_instance.global_vertex,graph_instance.gpu_vertex,(graph_instance.vertex_size+2)*sizeof(graph::vertex),hipMemcpyDeviceToHost);
    
    printf("over\n");
    
    hipMemcpy(TEPS,TEPS_GPU,num_block*num_threads*sizeof(int),hipMemcpyDeviceToHost);
    hipFree(graph_instance.gpu_vertex);
    hipFree(graph_instance.gpu_edge);
    hipFree(graph_instance.gpu_used_result_buf);  
    hipFree(TEPS_GPU);
    free(temp_vertex_array);
    int total_Teps=0;
    for(int i=0; i<num_block*num_threads; ++i){
        total_Teps+=TEPS[i];
    }
    bellman_ford(graph_instance, total_Teps);
    return total_Teps;
}

void bellman_ford(graph &Graph, int &total_Teps){

    int *GPU_Distances;
    int totalThreads = 1024 * ceil(Graph.vertex_size/1024.0);
    hipMalloc((void **)&Graph.gpu_vertex,Graph.vertex_size*sizeof(graph::vertex));
    hipMalloc((void **)&Graph.gpu_edge,Graph.edges_size*sizeof(graph::edge));
    hipMalloc((void **)&GPU_Distances,Graph.vertex_size*sizeof(int));
    hipMemcpy(Graph.gpu_vertex,Graph.global_vertex,Graph.vertex_size*sizeof(graph::vertex),hipMemcpyHostToDevice);
    hipMemcpy(Graph.gpu_edge,Graph.global_edge,Graph.edges_size*sizeof(graph::edge),hipMemcpyHostToDevice);
    int *distances = new int[Graph.vertex_size];
    int *TEPS = new int[totalThreads], *GPU_TEPS;
    for(int i=0; i<totalThreads; ++i)
        TEPS[i]=0;
    hipMalloc((void **)&GPU_TEPS, sizeof(int) * totalThreads);
    hipMemcpy(GPU_TEPS,TEPS,sizeof(int) * totalThreads, hipMemcpyHostToDevice);
    for(int i=0; i<Graph.vertex_size; ++i){
        distances[i]=Graph.global_vertex[i].dist;
    }
    hipMemcpy(GPU_Distances,distances,Graph.vertex_size*sizeof(int),hipMemcpyHostToDevice);
    bool *change = new bool, *GPU_change;
    *change=true;
    hipMalloc((void **) &GPU_change, sizeof(bool));
    while(*change){
        *change=false;
        bellmanFordKuda<<<dim3(ceil(Graph.vertex_size/1024.0),1,1),dim3(1024,1,1)>>>(Graph.gpu_vertex,Graph.gpu_edge,Graph.edges_size,Graph.vertex_size,GPU_Distances,GPU_change,GPU_TEPS);
        hipMemcpy(change,GPU_change,sizeof(bool),hipMemcpyDeviceToHost);
    }
    hipMemcpy(Graph.global_vertex,Graph.gpu_vertex,Graph.vertex_size*sizeof(graph::vertex),hipMemcpyDeviceToHost);
    //printResult(Graph);
    hipMemcpy(TEPS,GPU_TEPS,sizeof(int) * totalThreads, hipMemcpyDeviceToHost);
    for(int i=0; i<totalThreads; ++i)
        total_Teps+=TEPS[i];
    hipFree(Graph.gpu_edge);
    hipFree(Graph.gpu_vertex);
    hipFree(GPU_Distances);
    hipFree(GPU_change);
    hipFree(GPU_TEPS);
    free(distances);
}