#include "hip/hip_runtime.h"
#ifndef _RELAX_H_
#define _RELAX_H_
#include "vertex.cu"
__global__ 
void get_result(graph::vertex* gpu_global_vertex,int i){
	printf("result: %d\n",gpu_global_vertex[i].dist);	
}

__global__ 
void relax_all(int* gpu_vertex_buf, graph::vertex* gpu_global_vertex, graph::edge* gpu_global_edge, graph::gpuResult* cpu_result, graph::gpuResult* gpu_used_result_buf, int* TEPS_GPU){
    const unsigned int bid = blockIdx.x; 
    const unsigned int num_block = gridDim.x; 
    const unsigned int tid_in_block = threadIdx.x;
    const unsigned int num_thread = blockDim.x;
    const unsigned int tid_in_grid = blockDim.x * blockIdx.x +threadIdx.x;
    //printf("thread id: %d, thread teps: %d\n",tid_in_grid,TEPS_GPU[tid_in_grid]);

    int i=0,j=0;
    for (i=bid;i<MAX_BUCKET_SIZE;i+=num_block){
        graph::vertex *temp_v = &gpu_global_vertex[gpu_vertex_buf[i]];
        int num_edges = gpu_global_vertex[gpu_vertex_buf[i]+1].edge_index - temp_v->edge_index;
        int tent_current = temp_v->dist;
        if(gpu_vertex_buf[i] == 0)
            return;   
        for(j=tid_in_block;j<MAX_RESULT_SIZE;j+=num_thread){
            int dist_current = 0;
            int dest = 0;
            int tent_dest = 0;
            int flag = 0;
            if(j < num_edges){
                TEPS_GPU[tid_in_grid]++;
                dist_current = gpu_global_edge[temp_v->edge_index+j].distance;
                dest = gpu_global_edge[temp_v->edge_index+j].des_v;
                tent_dest = gpu_global_vertex[dest].dist;
            }
            if(tent_current + dist_current < tent_dest){
                gpu_global_vertex[dest].dist = tent_current + dist_current;
                gpu_global_vertex[dest].pre_vertex = i;
                flag =1;
            }

            gpu_used_result_buf[j+32*bid].index = dest*flag;
                    gpu_used_result_buf[j+32*bid].old_distance = tent_dest*flag;
                    gpu_used_result_buf[j+32*bid].new_distance = (tent_current+dist_current)*flag;
            if(dest*flag==1275){
                printf("@@@%d %d\n",j+32*bid,gpu_used_result_buf[j+32*bid].index);
                gpu_used_result_buf[j+32*bid].index = dest*flag;
            }
        }
    }
}

__global__
void bellmanFordKuda(graph::vertex* gpu_vertex, graph::edge* gpu_edge, int edges_size, int vertex_size, int* distances, bool *change, int* TEPS){
    *change=false;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<vertex_size){
        
        int source=0;
        for(int j=0; j<edges_size; ++j){
            if(source<vertex_size-1 && j==gpu_vertex[source+1].edge_index)
                ++source;
                graph::edge edge = gpu_edge[j];
            if(edge.des_v==i){
                TEPS[i]++;
                int aux=distances[source]+edge.distance;
                if(aux<gpu_vertex[i].dist){
                    gpu_vertex[i].dist=aux;
                    gpu_vertex[i].pre_vertex=source;
                    *change=true;
                }
            }
        }
        for(int i=0; i<vertex_size; ++i)
            distances[i]=gpu_vertex[i].dist;
    }
}

#endif
