#include "vertex.h"

graph::graph(char* filepath, int D){
    init_memory(filepath);
    delta = D;
    gpu_result_buf = (graph::gpuResult*)malloc(MAX_RESULT_SIZE * sizeof(graph::gpuResult));
    printf("end of graph init\n");
    src = 1;
    global_vertex[src].dist = 0;
    dest = 4;
    init_all_bucket();
}

graph::~graph(){
    free(global_vertex);
    free(global_edge);
    free(gpu_result_buf);
}

int graph::init_graph(){
    int i;
   
    global_vertex =(struct vertex*) malloc((vertex_size+2)*sizeof(struct vertex));
    global_edge = (struct edge*)malloc(edges_size*sizeof(struct edge));
  
   
    for(i=0;i<vertex_size+2;i++){
      global_vertex[i].edge_index =0;
      global_vertex[i].dist = MAX_DISTANCE;
      global_vertex[i].pre_vertex = -1;
    }
  
    graph_init=1;
    return 0;
  }

int graph::init_memory(char* filepath){
    char string[256];
  
    FILE* fp = fopen(filepath,"r");
    if(fp==NULL)
      return -1;
  
    
    while(fgets(string,256,fp)!=NULL){
      static char sign;
    
      static int src,dest,dist,cur_v=0,cur_edge=0;
    
      sscanf(string,"%c",&sign);
  
    
      if(sign=='a'){
          if(!graph_init)
              return -2;
          if(cur_edge>edges_size)
              return -4;
        sscanf(string,"%c\t%d\t%d\t%d",&sign,&src,&dest,&dist);
    
    
        global_edge[cur_edge].des_v=dest;
        global_edge[cur_edge].distance=dist;
        cur_edge++;
  
  
        if(cur_v!=src){
    
          if(cur_v==src-1){
    
              global_vertex[src].edge_index=cur_edge-1;	
              cur_v=src;
          }
          else
              return -3;
        }
      }
      //the line describe the size of graph
      else if(sign=='p'){ 
        sscanf(string,"%c\t%d\t%d",&sign,&src,&dest);
        vertex_size = src;
        edges_size = dest;
        printf("GOT the size of graph, vertex:%d edge:%d\n",vertex_size,edges_size);
        init_graph();
      }
    }
  
    fclose(fp);
    printf("end of init\n");
    //copy to GPU
   /* CUDA_SAFE_CALL(hipMalloc((void **)&gpu_vertex,(vertex_size+2)*sizeof(struct vertex)));
    CUDA_SAFE_CALL(hipMalloc((void**)&gpu_edge,edges_size*sizeof(struct edge)));
    CUDA_SAFE_CALL(hipMemcpy(gpu_vertex,global_vertex,(vertex_size+2)*sizeof(struct vertex),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gpu_edge,global_edge,edges_size*sizeof(struct edge)));
    */
    return 0;
  }

int graph::is_all_bucket_empty(){
    return min_no_empty_bucket()==-1;
}

int graph::min_no_empty_bucket(){
    for(int i=0;i<MAX_BUKET_NUM;i++){
        if(!bucket_array[i].empty()){
              return i;
        }
    }
    return -1;
}

int graph::bucket_set_to_array(int index, int* array){
    int count = 0;
    std::set<int>::iterator it = bucket_array[index].begin();
    for(;it!=bucket_array[index].end();it++){
            array[count]=*it;
            count++;

	    if(count>=8)
		return 8;
	    if(index==62){
	    printf("!!! %d\n",*it);
}
	    if(*it == 1354){
	    printf("%d oooops!1354\n",index);
}
	    if(count>MAX_BUCKET_SIZE){
		printf("oops!\n");
		exit(1);
	    }
        }
    return count;
}

int graph::init_all_bucket(){
    printf("insert src : %d\n", src);
    bucket_array[0].insert(src);
    return 1;
}

int graph::getFinishedNodes(int bucket){
    int result=0;
    for(int i=0; i<bucket; ++i){
        result+=bucket_array[i].size();
    }
    return result;
}