#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <chrono>

#define TILE_WIDTH 32

typedef int dato;

using namespace std;
using namespace std::chrono;

template<class T>
void printMatrix(T *M, int rows, int cols) {
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			cout << M[i*cols + j] << '\t';
		}
		cout << endl;
	}
	cout << endl;
}

__global__
void matrixMulKernel(dato *M, dato *N, dato *P, int a, int b, int c) {
	int col = blockIdx.y*blockDim.y + threadIdx.y;
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	if (row < a && col < c) {
		dato Pvalue = 0;
		for (int k = 0; k < b; ++k) {
			Pvalue += M[row*b + k] * N[k*c + col];
		}
		P[row*c + col] = Pvalue;
	}
}

__global__
void matrixMulKernel2(dato *M, dato *N, dato *P, int a, int b, int c) {
	__shared__ dato Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ dato Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x, 
		by = blockIdx.y, 
		tx = threadIdx.x, 
		ty = threadIdx.y;
	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;
	dato pValue = 0;
	for (int ph = 0; ph < ceil(b / (float)TILE_WIDTH); ++ph) {
		if (row < a && (ph*TILE_WIDTH + tx) < b)
			Mds[ty][tx] = M[row*b + ph * TILE_WIDTH + tx];
		else
			Mds[ty][tx] = 0;
		if (col < c && (ph*TILE_WIDTH + ty) < b)
			Nds[ty][tx] = N[(ph*TILE_WIDTH + ty)*c + col];
		else
			Nds[ty][tx] = 0;
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k) {
			pValue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	if(row<a && col<c)
		P[row*c + col] = pValue;
}

__global__
void matrixMulKernel3(dato *M, dato *N, dato *P, int a, int b, int c) {
	__shared__ dato Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ dato Nds[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x,
		by = blockIdx.y,
		tx = threadIdx.x,
		ty = threadIdx.y;
	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;
	dato pValue = 0;
	for (int ph = 0; ph < ceil(b / (float)TILE_WIDTH); ph+=2) {
		if (row < a && (ph*TILE_WIDTH + tx) < b)
			Mds[ty][tx] = M[row*b + ph * TILE_WIDTH + tx];
		else
			Mds[ty][tx] = 0;
		if (col < c && (ph*TILE_WIDTH + ty) < b)
			Nds[ty][tx] = N[(ph*TILE_WIDTH + ty)*c + col];
		else
			Nds[ty][tx] = 0;
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k) {
			pValue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	if (row < a && col < c)
		P[row*c + col] = pValue;
}

void matrixMul(dato *M, dato *N, dato *P, int a, int b, int c, int mode) {
	dato *D_M, *D_N, *D_P;
	int matrixSize1 = a * b;
	int matrixSize2 = b * c;
	int matrixSize3 = a * c;
	hipMalloc((void**)&D_M, matrixSize1 * sizeof(dato));
	hipMemcpy(D_M, M, matrixSize1 * sizeof(dato), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_N, matrixSize2 * sizeof(dato));
	hipMemcpy(D_N, N, matrixSize2 * sizeof(dato), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_P, matrixSize3 * sizeof(dato));
	auto start = high_resolution_clock::now();
	switch (mode) {
	case 0:
		matrixMulKernel << <dim3(ceil(a / 32.0), ceil(c / 32.0), 1), dim3(32, 32, 1) >> > (D_M, D_N, D_P, a, b, c);
		break;
	case 1:
		matrixMulKernel2 << <dim3(ceil(a / 32.0), ceil(c / 32.0), 1), dim3(32, 32, 1) >> > (D_M, D_N, D_P, a, b, c);
		break;
	case 2:
		matrixMulKernel3 << <dim3(ceil(a / 32.0), ceil(c / 32.0), 1), dim3(32, 32, 1) >> > (D_M, D_N, D_P, a, b, c);
	}
	auto end = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(end - start);
	cout <<"Tiempo en microsegundos: "<< duration.count() << endl;

	hipMemcpy(P,D_P,matrixSize3*sizeof(dato),hipMemcpyDeviceToHost);

	hipFree(D_M);
	hipFree(D_N);
	hipFree(D_P);
}

int main(){
	//srand(time(NULL));
	dato *M1, *M2, *M3;
	int a=1000, b=1000, c=1000;
	M1 = new dato[a*b];
	M2 = new dato[b*c];
	M3 = new dato[a*c];
	for (int i = 0, top = a * b; i < top; ++i) {
		M1[i] = rand()%5;
	}
	for (int i = 0, top = b * c; i < top; ++i) {
		M2[i] = rand()%5;
	}
	//printMatrix(M1, a, b);
	//printMatrix(M2, b, c);
	matrixMul(M1, M2, M3, a, b, c, 2);
	printMatrix(M3, a, c);
	//matrixMul(M1, M2, M3, a, b, c, 1);
	//printMatrix(M3, a, c);
	//matrixMul(M1, M2, M3, a, b, c, 0);
    return 0;
}