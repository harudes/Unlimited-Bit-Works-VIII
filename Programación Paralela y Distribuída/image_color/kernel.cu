#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <opencv2/opencv.hpp>
#include <iostream>

using namespace cv;
using namespace std;

#define CHANNELS 3
#define BLUR_SIZE 6

__global__
void colorToGreyscaleConversion(unsigned char * Pout, unsigned	char * Pin, int width, int height) {
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	if (Col < width && Row < height) {
		int greyOffset = Row * width + Col;
		int rgbOffset = greyOffset * CHANNELS;
		unsigned char r = Pin[rgbOffset];
		unsigned char g = Pin[rgbOffset + 1];
		unsigned char b = Pin[rgbOffset + 2];
		Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}

void gray(unsigned char *Pin, unsigned char *Pout, int width, int height) {
	uchar *d_pin;
	uchar *d_pout;
	hipMalloc((void**)&d_pin, width*height * 3);
	hipMalloc((void**)&d_pout, width*height);

	hipMemcpy(d_pin, Pin, width*height*3,hipMemcpyHostToDevice);
	
	colorToGreyscaleConversion <<< dim3(ceil(width / 32.0), ceil(height / 32.0), 1), dim3(32, 32, 1) >>> (d_pout, d_pin, width, height);

	hipMemcpy(Pout, d_pout, width*height, hipMemcpyDeviceToHost);

	hipFree(d_pin);
	hipFree(d_pout);
}

__global__
void blurKernel(unsigned char *in, unsigned char *out, int w, int h) {
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	if (Col < w && Row < h) {
		for (int i = 0; i < 3; ++i) {
			int pixVal = 0;
			int pixels = 0;
			for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
				for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
					int curRow = Row + blurRow;
					int curCol = Col + blurCol;
					if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
						pixVal += in[(curRow * w + curCol)*3 + i];
						pixels++;
					}
				}
			}
			out[(Row * w + Col)*3 + i] = (unsigned char)(pixVal / pixels);
		}
	}
}

void blur(unsigned char *in, unsigned char *out, int w, int h) {
	uchar *d_in;
	uchar *d_out;
	hipMalloc((void**)&d_in, w*h * 3);
	hipMalloc((void**)&d_out, w*h * 3);

	hipMemcpy(d_in, in, w*h * 3, hipMemcpyHostToDevice);

	blurKernel <<< dim3(ceil(w / 32.0), ceil(h / 32.0), 1), dim3(32, 32, 1) >>> (d_in, d_out, w, h);

	hipMemcpy(out, d_out, w*h*3, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}

void maingray(string image) {
	Mat img = imread(image, IMREAD_COLOR);
	namedWindow("image");
	imshow("image", img);
	int imgSize = img.total()*img.channels();
	uchar *Pin = img.isContinuous() ? img.data : img.clone().data;
	int newImgSize = img.total();
	uchar *Pout = new uchar[newImgSize];

	gray(Pin, Pout, img.cols, img.rows);

	Mat newImg(img.rows, img.cols, CV_8UC1, Pout, Mat::AUTO_STEP);
	imwrite("gray_" + image, newImg);
	namedWindow("image2");
	imshow("image2", newImg);
	waitKey(0);
}

void mainblur(string image) {
	Mat img = imread(image, IMREAD_COLOR);
	namedWindow("image");
	imshow("image", img);
	int imgSize = img.total()*img.channels();
	uchar *Pin = img.isContinuous() ? img.data : img.clone().data;
	uchar *Pout = new uchar[imgSize];

	blur(Pin, Pout, img.cols, img.rows);

	Mat newImg(img.rows, img.cols, CV_8UC3, Pout, Mat::AUTO_STEP);
	imwrite("blur_" + image, newImg);
	namedWindow("image2");
	imshow("image2", newImg);
	waitKey(0);
}

int main(int argc, char** argv)
{
	//maingray("dmc5.jpg");
	mainblur("dmc5.jpg");
}