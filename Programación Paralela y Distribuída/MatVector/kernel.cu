
#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>

using namespace std;

__global__
void matVecKernel(float* mat, float* vec, float* result, int m) {
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	if (row < m) {
		for (int i = 0; i < m; ++i) {
			result[row] += mat[row*m + i]*vec[i];
		}
	}
}

void matVec(float* mat, float* vec, float* result, int m) {
	float *D_mat, *D_vec, *D_result;
	hipMalloc((void**)&D_mat,m*m *sizeof(float));
	hipMemcpy(D_mat,mat,m*m * sizeof(float),hipMemcpyHostToDevice);
	hipMalloc((void**)&D_vec, m * sizeof(float));
	hipMemcpy(D_vec, vec, m * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_result, m * sizeof(float));

	matVecKernel <<< ceil(m / 1024.0), 1024 >>> (D_mat, D_vec, D_result, m);

	hipMemcpy(result, D_result, m * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(D_mat);
	hipFree(D_vec);
	hipFree(D_result);
}

int main(){
	int m;
	m = 1000;
	float *mat = new float[m*m], *vec = new float[m], *result = new float[m];
	for (int i = 0; i < m*m; ++i)
		mat[i] = i;
	for (int i = 0; i < m; ++i) {
		vec[i] = 2;
		result[i] = 0;
	}
	matVec(mat,vec,result,m);
	/*for (int i = 0; i < m; ++i)
		cout << result[i] << "\t";
	cout << endl;*/
    return 0;
}
