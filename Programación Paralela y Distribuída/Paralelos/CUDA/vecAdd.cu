#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

// cuda: usr local cuda bin nvcc

#define vecSize 10000

using namespace std;

__global__
void vecAddKernel(float *A, float *B, float *C, int n){
    int i= blockDim.x * blockIdx.x+ threadIdx.x;
    if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* h_A, float* h_B, float* h_C, int n){
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C; 

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_C, size);

    vecAddKernel<<<ceil(n/256.0),256>>>(d_A,d_B,d_C,n);

    hipMemcpy(h_C,d_C,size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(){
    float a[vecSize], b[vecSize],c[vecSize];
    for(int i=0; i<vecSize; ++i){
        a[i]=i*2;
        b[i]=i*3;
    }
    vecAdd(a,b,c,vecSize);
    for(int i=0; i<vecSize; ++i)
        cout<<c[i]<<" ";
    cout<<endl;
    int dev_count;
    hipGetDeviceCount(&dev_count);
    cout<<dev_count<<endl;
    return 0;
}