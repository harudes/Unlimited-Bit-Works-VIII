#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__
void vecAddKernel(float* A, float* B, float* C, int n){
	printf("-A: %f B: %f ",*A,*B);
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<n) C[i] = A[i] + B[i];
	printf("C: %f \n", *C);
}
void vecAdd(float* A, float* B, float* C, int n){
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	
	hipMalloc((void**) &d_A,size);
	hipMemcpy(d_A, A,size,hipMemcpyHostToDevice);
	hipMalloc((void**) &d_B,size);
	hipMemcpy(d_B, B,size,hipMemcpyHostToDevice);
	
	hipMalloc((void**) &d_C,size);
	//vecAddKernel<<<ceil(n/256.0),256>>>(d_A,d_B,d_C,n);
	vecAddKernel<<<1,10>>>(d_A,d_B,d_C,n);
	hipMemcpy(C, d_C,size, hipMemcpyDeviceToHost);
	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	printf("d_c: %f \n", *C);
}

int main(){
	printf("nani: \n");
	float A[10];
	float B[10];	
	for(int i=0;i<10;++i){
		A[i] = 1.0;
		B[i] = 2.0;
	}
	float C[10];
	vecAdd(A,B,C,10);
	for(int i=0;i<10;++i){
		printf("%f ",C[i]);
	}
	printf("\n");
	return 0;
}

/**usr/local/cuda/bin*/
