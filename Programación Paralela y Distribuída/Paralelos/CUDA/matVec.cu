#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

using namespace std;
using namespace std::chrono;

__global__
void matVecKernel(float* mat, float* vec, float* result, int m) {
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	if (row < m) {
		for (int i = 0; i < m; ++i) {
			result[row] += mat[row*m + i]*vec[i];
		}
	}
}

void matVec(float* mat, float* vec, float* result, int m) {
	float *D_mat, *D_vec, *D_result;
	hipMalloc((void**)&D_mat,m*m *sizeof(float));
	hipMemcpy(D_mat,mat,m*m * sizeof(float),hipMemcpyHostToDevice);
	hipMalloc((void**)&D_vec, m * sizeof(float));
	hipMemcpy(D_vec, vec, m * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_result, m * sizeof(float));

	auto start = high_resolution_clock::now();
	matVecKernel <<< ceil(m / 1024.0), 1024 >>> (D_mat, D_vec, D_result, m);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<nanoseconds>(stop - start);
	cout<<"Tiempo kernel: "<< duration.count()<<" nanosegundos"<<endl;

	hipMemcpy(result, D_result, m * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(D_mat);
	hipFree(D_vec);
	hipFree(D_result);
}

int main(){
	cout<<"Multiplicacion matriz vector"<<endl;
	int m;
	m = 1000;
	cout<<"Matriz de dimension "<<m<<"x"<<m<<endl;
	float *mat = new float[m*m], *vec = new float[m], *result = new float[m];
	for (int i = 0; i < m*m; ++i)
		mat[i] = i;
	for (int i = 0; i < m; ++i) {
		vec[i] = 2;
		result[i] = 0;
	}
	auto start = high_resolution_clock::now();
	matVec(mat,vec,result,m);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<nanoseconds>(stop - start);
	cout<<"Tiempo total: "<< duration.count()<<" nanosegundos"<<endl;
	
	/*for (int i = 0; i < m; ++i)
		cout << result[i] << "\t";
	cout << endl;*/
    return 0;
}
