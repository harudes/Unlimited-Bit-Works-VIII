
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__
void matrixSumKernel1(float *mat1, float *mat2, float *mat3, int m, int n) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	if (col < n && row < m) {
		mat3[row*n + col] = mat1[row*n + col] + mat2[row*n + col];
	}
}

__global__
void matrixSumKernel2(float *mat1, float *mat2, float* mat3, int m, int n) {
	int row = threadIdx.x + blockIdx.x * blockDim.x;
	if (row < m) {
		for (int i = 0; i < n; ++i) {
			mat3[row*n+i] = mat1[row*n + i] + mat2[row*n + i];
		}
	}
}

__global__
void matrixSumKernel3(float *mat1, float *mat2, float* mat3, int m, int n) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	if (col < n) {
		for (int i = 0; i < m; ++i) {
			mat3[i*n + col] = mat1[i*n + col] + mat2[i*n + col];
		}
	}
}

void matrixSum(float* mat1, float* mat2, float* mat3, int m, int n, int mode) {
	if (mode <= 2 && mode >= 0) {
		int matSize = m * n;
		float *D_mat1, *D_mat2, *D_mat3;
		hipMalloc((void **)&D_mat1, matSize*sizeof(float));
		hipMemcpy(D_mat1, mat1, matSize * sizeof(float), hipMemcpyHostToDevice);
		hipMalloc((void **)&D_mat2, matSize * sizeof(float));
		hipMemcpy(D_mat2, mat2, matSize * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void **)&D_mat3, matSize * sizeof(float));
        auto start = high_resolution_clock::now();
		switch (mode) {
		case 0:
			matrixSumKernel1 <<< dim3(ceil(n / 32.0), ceil(m / 32.0), 1), dim3(32, 32, 1) >>> (D_mat1, D_mat2, D_mat3, m, n);
			break;
		case 1:
			matrixSumKernel2 <<< ceil(m / 1024.0), 1024 >>> (D_mat1, D_mat2, D_mat3, m, n);
			break;
		case 2:
			matrixSumKernel3 <<< ceil(n / 1024.0), 1024 >>> (D_mat1, D_mat2, D_mat3, m, n);
			break;
		}
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<nanoseconds>(stop - start);
		cout<<"Tiempo kernel: "<< duration.count()<<" nanosegundos"<<endl;
        
		hipMemcpy(mat3, D_mat3, matSize * sizeof(float), hipMemcpyDeviceToHost);
		hipFree(D_mat1);
		hipFree(D_mat2);
		hipFree(D_mat3);
	}
}

int main(int argc, char* argv[]){
	cout<<"Suma de matrices"<<endl;
	int m, n;
	m = 1000;
	n = 1500;
	cout<<"Matrices de dimension "<<m<<"x"<<n<<endl;
	float *mat1 = new float[m*n], *mat2 = new float[m*n], *mat3 = new float[m*n];
	for (int i = 0, size = m*n; i < size; ++i) {
		mat1[i] = i;
		mat2[i] = size - i;
	}
	for(int i=0; i<3; ++i){
		auto start = high_resolution_clock::now();
		matrixSum(mat1, mat2, mat3, m, n, i);
		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<nanoseconds>(stop - start);
		cout<<"Tiempo total: "<< duration.count()<<" nanosegundos con el metodo "<<i+1<<endl;
	}
	/*for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			cout << mat3[i*n+j] << " ";
		}
		cout << endl;
	}
	cout << endl;*/
    return 0;
}
