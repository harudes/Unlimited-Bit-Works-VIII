#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define CHANNELS 3
using namespace std;

__global__
void colorToGreyscaleConversion(unsigned char *Pout, unsigned char *Pin, int width, int height){
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    if(Col < width && Row < height){
        int greyOffset = Row*width + col;
        int rgbOffset = greyOffset * CHANNELS;
        unsigned char r = Pin[rgbOffset];
        unsigned char g = Pin[rgbOffset + 1];
        unsigned char b = Pin[rgbOffset + 2];
        Pout[grayOffset] = 0.21f* r + 0.71f * g + 0.07f * b;
    }
}

int main(int argc, char* argv[]){
    return 0;
}